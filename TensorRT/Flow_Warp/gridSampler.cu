#include "hip/hip_runtime.h"
#include "gridSampler.h"
#include "gridSampler.cuh"
#include <stdio.h>
#include <assert.h>

using half = __half;

// alignd with torch Macros.h
// CUDA_MAX_THREADS_PER_BLOCK is same for all architectures currently
constexpr unsigned int CUDA_MAX_THREADS_PER_BLOCK = 1024;
// CUDA_THREADS_PER_BLOCK_FALLBACK is the "canonical fallback" choice of block size.
// 256 is a good number for this fallback and should give good occupancy and
// versatility across all architectures.
constexpr unsigned int CUDA_THREADS_PER_BLOCK_FALLBACK = 256;

#define C10_MAX_THREADS_PER_BLOCK(val) (((val) <= CUDA_MAX_THREADS_PER_BLOCK) ? (val) : CUDA_THREADS_PER_BLOCK_FALLBACK)
#define C10_LAUNCH_BOUNDS_1(max_threads_per_block) __launch_bounds__((C10_MAX_THREADS_PER_BLOCK((max_threads_per_block))))

// aligned with KernelUtils.h
#define CUDA_KERNEL_LOOP(i, n) \
  int64_t _i_n_d_e_x = blockIdx.x * blockDim.x + threadIdx.x;                                \
  for (int i=_i_n_d_e_x; _i_n_d_e_x < (n); _i_n_d_e_x+=blockDim.x * gridDim.x, i=_i_n_d_e_x)


constexpr int CUDA_NUM_THREADS = 1024;
inline int GET_BLOCKS(const int N)
{
  assert(N > 0);
  return (N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS;
}

template <typename scalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void grid_sampler_2d_kernel(
    int nthreads,
    const scalar_t* inputPtr,
    const scalar_t* gridPtr,      
    scalar_t* const outputPtr,      
    int C,
    int inp_H,
    int inp_W,
    int out_H, // same as grid_H
    int out_W, // same as grid_W
    int inp_sN,
    int inp_sC,
    int inp_sH,
    int inp_sW,
    int grid_sN,//gridHeight*gridWidth*2
    int grid_sH, // gridWidth*gridHeight
    int grid_sW,// gridWidth
    int grid_sCoor,//1
    int out_sN, // gridWidth*gridHeight*InputChannel
    int out_sC, // gridWidth*gridHeight
    int out_sH, // gridWidth
    int out_sW, //1
    torch::detail::GridSamplerInterpolation interpolation_mode,
    torch::detail::GridSamplerPadding padding_mode,
    bool align_corners) {
// 输入参数 第一个顾名思义 后三个为 输入（input和grid） 输出指针
//紧接着四个为通道数输入 输出的 H W size
//inp_sN 及下面三个为  input的 NCHW 排序为了获取到 第几个N 几个C ....  其中inp_sN=C*inp_H*inp_W; inp_sC = inp_H*inp_W; inp_sH=inp_W; inp_sW=1;
//grid_sN 及下面三个为 grid的排序 不input不同 顺序为 NHW2 计算方法通input，
//同理out_sN yu inp_sN 类似
// 差值模式和padd模式  在头文件中用了枚举进行了定义
//nchw 与线性排列之间关系。linear_index = (n * C * H * W) + (c * H * W) + (h * W) + w
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int w = index % out_W;
    const int h = (index / out_W) % out_H;
    const int n = index / (out_H * out_W); // n是batchSize \
    
    const int grid_offset = n * grid_sN + h * grid_sW + w ;

    // get the corresponding input x, y co-ordinates from grid
    scalar_t ix = gridPtr[grid_offset];
    scalar_t iy = gridPtr[grid_offset + grid_sH];

    ix = ix + w;
    iy = iy + h;
    //ix = grid_sampler_compute_source_index(ix, inp_W, padding_mode, align_corners);
    //iy = grid_sampler_compute_source_index(iy, inp_H, padding_mode, align_corners);

    if (interpolation_mode == torch::detail::GridSamplerInterpolation::Bilinear) {
      // get NE, NW, SE, SW pixel values from (x, y) 
      int ix_nw = static_cast<int>(floorf(ix)); //floorf 返回不大于 ix 的最大整数值
      int iy_nw = static_cast<int>(floorf(iy));  //北为N 南为S 西为 W 东为E
      int ix_ne = ix_nw + 1;                    // nw 为西北左上角 sw 为西南左下角 ne为东北右上角 se为东南 右下角
      int iy_ne = iy_nw;
      int ix_sw = ix_nw;
      int iy_sw = iy_nw + 1;
      int ix_se = ix_nw + 1;
      int iy_se = iy_nw + 1;

      // get surfaces to each neighbor:
      scalar_t nw = (static_cast<scalar_t>(ix_se) - ix)    * (static_cast<scalar_t>(iy_se) - iy);
      scalar_t ne = (ix    - ix_sw) * (static_cast<scalar_t>(iy_sw) - iy);
      scalar_t sw = (static_cast<scalar_t>(ix_ne) - ix)    * (iy    - iy_ne);
      scalar_t se = (ix    - ix_nw) * (iy    - iy_nw);
      //  计算该四个值为 双线性插值 化简后公式 可参考 https://www.cnblogs.com/wancy/p/15212604.html 最后化简后的公式

      // calculate bilinear weighted pixel value and set output pixel
      auto inp_ptr_NC = inputPtr + n * inp_sN;
      auto out_ptr_NCHW = outputPtr + n * out_sN + h * out_sH + w * out_sW;
      for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
        *out_ptr_NCHW = static_cast<scalar_t>(0);
        if (within_bounds_2d(iy_nw, ix_nw, inp_H, inp_W)) {
          *out_ptr_NCHW += inp_ptr_NC[iy_nw * inp_sH + ix_nw * inp_sW] * nw;
        }
        if (within_bounds_2d(iy_ne, ix_ne, inp_H, inp_W)) {
          *out_ptr_NCHW += inp_ptr_NC[iy_ne * inp_sH + ix_ne * inp_sW] * ne;
        }
        if (within_bounds_2d(iy_sw, ix_sw, inp_H, inp_W)) {
          *out_ptr_NCHW += inp_ptr_NC[iy_sw * inp_sH + ix_sw * inp_sW] * sw;
        }
        if (within_bounds_2d(iy_se, ix_se, inp_H, inp_W)) {
          *out_ptr_NCHW += inp_ptr_NC[iy_se * inp_sH + ix_se * inp_sW] * se;
        }
      }
    } else if (interpolation_mode == torch::detail::GridSamplerInterpolation::Nearest) {
      int ix_nearest = static_cast<int>(roundf(ix));
      int iy_nearest = static_cast<int>(roundf(iy));

      // assign nearest neighor pixel value to output pixel
      auto inp_ptr_NC = inputPtr + n * inp_sN;
      auto out_ptr_NCHW = outputPtr + n * out_sN + h * out_sH + w * out_sW;
      for (int c = 0; c < C; ++c, inp_ptr_NC += inp_sC, out_ptr_NCHW += out_sC) {
        if (within_bounds_2d(iy_nearest, ix_nearest, inp_H, inp_W)) {
          *out_ptr_NCHW = inp_ptr_NC[iy_nearest * inp_sH + ix_nearest * inp_sW];
        } else {
          *out_ptr_NCHW = static_cast<scalar_t>(0);
        }
      }

    }
  }
}

// No shape checking needed here. See # NOTE [ grid_sampler Native Functions ].
int grid_sampler_2d_cuda(int batchSize, const void* inputPtr, const void* gridPtr,
  void* const outputPtr,
  int C,
  int inp_H,
  int inp_W,
  int out_H, // same as grid_H
  int out_W, // same as grid_W
  int inp_sN,
  int inp_sC,
  int inp_sH,
  int inp_sW,
  int grid_sN,
  int grid_sH,
  int grid_sW,
  int grid_sCoor,
  int out_sN,
  int out_sC,
  int out_sH,
  int out_sW,
  torch::detail::GridSamplerInterpolation interpolation_mode,
  torch::detail::GridSamplerPadding padding_mode,
  bool align_corners, torch::detail::GridSamplerDataType dataType, hipStream_t stream)
{

  int count = out_H * out_W * batchSize; // will have inner loop over C
  if (count > 0) {
    if (dataType == torch::detail::GridSamplerDataType::GHALF)
    {
      grid_sampler_2d_kernel<half>
        <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(
          count,
          static_cast<const half*>(inputPtr), static_cast<const half*>(gridPtr),
          static_cast<half*>(outputPtr),
          C, inp_H, inp_W, out_H, out_W, inp_sN, inp_sC, inp_sH, inp_sW,
          grid_sN, grid_sH, grid_sW, grid_sCoor, out_sN, out_sC, out_sH, out_sW,
          interpolation_mode,
          padding_mode,
          align_corners);
    }
    else
    {
      grid_sampler_2d_kernel<float>
        <<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, stream>>>(
          count,
          static_cast<const float*>(inputPtr), static_cast<const float*>(gridPtr),
          static_cast<float*>(outputPtr),
          C, inp_H, inp_W, out_H, out_W, inp_sN, inp_sC, inp_sH, inp_sW,
          grid_sN, grid_sH, grid_sW, grid_sCoor, out_sN, out_sC, out_sH, out_sW,
          interpolation_mode,
          padding_mode,
          align_corners);
    }
  }

  return hipGetLastError() != hipSuccess;
}

